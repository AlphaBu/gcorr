

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <hipfft/hipfft.h>
#include <getopt.h>


int main(int argc, char *argv[]) {
  int nfft, batch, opt, tmp, ss, repeat;
  size_t nsample;
  float dtime;
  hipError_t status;
  hipfftReal *idata;
  hipfftComplex *odata, *icdata;
  hipfftResult result;
  hipfftHandle plan;
  hipEvent_t start_exec, stop_exec, start_fft, stop_fft;

  int ffts[]  = {4, 16,32,64,128,256,1024,4096,8192,16384};
  
  repeat = 10;
  nsample = 1024*1024*8;

  struct option options[] = {
    {"repeat", 1, 0, 'r'}, 
    {0, 0, 0, 0}
  };

  while (1) {
    opt=getopt_long_only(argc, argv, "n:r:", 
			 options, NULL);

    if (opt==EOF) break;

    switch (opt) {
      
    case 'r':
      ss = sscanf(optarg, "%d", &tmp);
      if (ss!=1)
        fprintf(stderr, "Bad -repeat option %s\n", optarg);
      else {
	repeat = tmp;
      }
      break; 

    case '?':
    default:
      break;
    }
  }

  hipEventCreate(&start_exec);
  hipEventCreate(&stop_exec);
  hipEventCreate(&start_fft);
  hipEventCreate(&stop_fft);

  // Start total time event
  hipEventRecord(start_exec, 0);


  // Allocate memory on the host
  status = hipMalloc(&idata, nsample*sizeof(hipfftReal));
  if (status != hipSuccess) {
    fprintf(stderr, "Error: hipMalloc failed (1)\n");
    return EXIT_FAILURE;
  }
  status = hipMalloc(&icdata, nsample*sizeof(hipfftComplex));
  if (status != hipSuccess) {
    fprintf(stderr, "Error: hipMalloc failed (2)\n");
    return EXIT_FAILURE;
  }
  status = hipMalloc(&odata, nsample*sizeof(hipfftComplex));
  if (status != hipSuccess) {
    fprintf(stderr, "Error: hipMalloc failed (3)\n");
    return EXIT_FAILURE;
  }

  printf("======= Real to Complex ========\n");
  printf("   n   |    time     |    1 GHz    |  Bandwidth |\n");

  int N = sizeof(ffts)/sizeof(int);
  for (int j=0; j<N; j++) {
    nfft = ffts[j]*2;

    // Setup the FFT
    batch = nsample/nfft;
    //result = cufftPlan1d(&plan, nfft, CUFFT_R2C, batch);
    result = hipfftPlanMany(&plan, 1, &nfft, NULL, 1, 0, NULL, 1, 0, HIPFFT_R2C, batch);
    if (result != HIPFFT_SUCCESS) {
      fprintf(stderr, "hipfftPlan1d failed with status %d\n", result);
      return EXIT_FAILURE;
    }
//    cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);

    hipEventRecord(start_fft, 0);
    for (int i=0; i<repeat; i++) {
      result = hipfftExecR2C(plan, idata, odata);
    }
    hipEventRecord(stop_fft, 0);
    hipEventSynchronize(stop_fft);
    hipEventElapsedTime(&dtime, start_fft, stop_fft);

    printf("%6d | %8.3f ms | %8.3f ms | %6.1f MHz |\n", nfft/2, dtime, dtime*4e9/((float)nfft*batch*repeat),
	   (float)nfft*batch*repeat/4/1e6/dtime*1000);

    // Destroy plan
    hipfftDestroy(plan);
  }

  printf("\n\n======= Complex to Complex ========\n");
  printf("   n   |    time     |    1 GHz    |  Bandwidth  |\n");

  N = sizeof(ffts)/sizeof(int);
  for (int j=0; j<N; j++) {
    nfft = ffts[j];

    // Setup the FFT
    batch = nsample/nfft;

    result = hipfftPlanMany(&plan, 1, &nfft, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);
    if (result != HIPFFT_SUCCESS) {
      fprintf(stderr, "hipfftPlanMany failed with status %d\n", result);
      return EXIT_FAILURE;
    }
//    cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE);

    hipEventRecord(start_fft, 0);
    for (int i=0; i<repeat; i++) {
      result = hipfftExecC2C(plan, icdata, odata, HIPFFT_FORWARD);
    }
    hipEventRecord(stop_fft, 0);
    hipEventSynchronize(stop_fft);
    hipEventElapsedTime(&dtime, start_fft, stop_fft);

    printf("%6d | %8.3f ms | %8.3f ms | %6.1f MHz |\n", nfft, dtime, dtime*2e9/((float)nfft*batch*repeat),
	   (float)nfft*batch*repeat/2/1e6/(dtime/1000));

    // Destroy plan
    hipfftDestroy(plan);

  }

  hipEventRecord(stop_exec, 0);
  hipEventSynchronize(stop_exec);
  hipEventElapsedTime(&dtime, start_exec, stop_exec);
  printf("\nTotal executution time =  %.3f ms\n", dtime);


  // Free allocated memory
  hipFree(idata);
  hipFree(icdata);
  hipFree(odata);

  hipEventDestroy(start_fft);
  hipEventDestroy(stop_fft);
  hipEventDestroy(start_exec);
  hipEventDestroy(stop_exec);
  

}
