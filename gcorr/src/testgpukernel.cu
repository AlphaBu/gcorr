#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <string>
#include <vector>
#include <cstdlib>
#include <cstring>
#include <stdint.h>
#include <complex>
#include <argp.h>

#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

#include "common.h"

#define NTHREADS 256

using std::string;
using std::cout;
using std::cerr;
using std::endl;
using std::vector;

const char *argp_program_version = "testgpukernel 1.0";
static char doc[] = "testgpukernel -- testing operation of the GPU correlator code";
static char args_doc[] = "configuration_file";

#define BUFSIZE 256

/* Our command line options */
static struct argp_option options[] = {
  { "loops", 'n', "NLOOPS", 0, "run the code N times in a loop" },
  { "binary", 'b', 0, 0, "output binary instead of default text" },
  { 0 }
};

struct arguments {
  int output_binary;
  int nloops;
  char configfile[BUFSIZE];
};

/* The option parser */
static error_t parse_opt(int key, char *arg, struct argp_state *state) {
  struct arguments *arguments = (struct arguments *)state->input;

  switch (key) {
  case 'b':
    arguments->output_binary = 1;
    break;
  case 'n':
    arguments->nloops = atoi(arg);
    break;
  case ARGP_KEY_END:
    if (strlen(arguments->configfile) == 0) {
      argp_usage(state);
      exit(0);
    }
    break;
  default:
    // Assume this is the config file.
    if (arg != NULL) {
       if (strlen(arg) > 0) {
       	  strncpy(arguments->configfile, arg, BUFSIZE);
       }
    }
  }
  return 0;
}

/* The argp parser */
static struct argp argp = { options, parse_opt, args_doc, doc };

int kNumStreams = 2;

#include "gxkernel.h"

void allocDataGPU(int8_t ****packedData, hipComplex ***unpackedData,
		  hipComplex ***channelisedData, hipComplex ***baselineData, 
		  float ***rotationPhaseInfo, float ***fractionalSampleDelays, int ***sampleShifts, 
      double ***gpuDelays, int numantenna, int subintsamples, int nbit, int nPol, bool iscomplex, int nchan, int numffts, int parallelAccum,
      int num_streams) {

  unsigned long long GPUalloc = 0;

  int packedBytes = subintsamples*nbit*nPol/8;
  *packedData = new int8_t**[num_streams];
  *unpackedData = new hipComplex*[num_streams];
  *channelisedData = new hipComplex*[num_streams];
  *baselineData = new hipComplex*[num_streams];
  *rotationPhaseInfo = new float*[num_streams];
  *fractionalSampleDelays = new float*[num_streams];
  *sampleShifts = new int*[num_streams];
  *gpuDelays = new double*[num_streams];

  // Unpacked data
  cout << "Alloc " << nchan*parallelAccum << " complex output values per baseline" << endl;
  for (int s=0; s<num_streams; s++) {
    (*packedData)[s] = new int8_t*[numantenna];
    for (int i=0; i<numantenna; i++) {
      gpuErrchk(hipMalloc(&((*packedData)[s])[i], packedBytes));
      GPUalloc += packedBytes;
    }

    gpuErrchk(hipMalloc(&(*unpackedData)[s], numantenna*nPol*subintsamples*sizeof(hipComplex)));
    GPUalloc += numantenna*nPol*subintsamples*sizeof(hipComplex);
  
    // FFT output
    gpuErrchk(hipMalloc(&(*channelisedData)[s], numantenna*nPol*subintsamples*sizeof(hipComplex)));
    GPUalloc += numantenna*nPol*subintsamples*sizeof(hipComplex);

    // Baseline visibilities
    int nbaseline = numantenna*(numantenna-1)/2;
    if (!iscomplex) subintsamples /= 2;
    gpuErrchk(hipMalloc(&(*baselineData)[s], nbaseline*4*nchan*parallelAccum*sizeof(hipComplex)));
    GPUalloc += nbaseline*4*nchan*parallelAccum*sizeof(hipComplex);

    // Fringe rotation vector (will contain starting phase and phase increment for every FFT of every antenna)
    gpuErrchk(hipMalloc(&(*rotationPhaseInfo)[s], numantenna*numffts*2*sizeof(float)));
    GPUalloc += numantenna*numffts*2*sizeof(float);

    // Fractional sample delay vector (will contain midpoint fractional sample delay [in units of radians per channel!] 
    // for every FFT of every antenna)
    gpuErrchk(hipMalloc(&(*fractionalSampleDelays)[s], numantenna*numffts*sizeof(float)));
    GPUalloc += numantenna*numffts*sizeof(float);

    // Sample shifts vector (will contain the integer sample shift relative to nominal FFT start for every FFT of every antenna)
    gpuErrchk(hipMalloc(&(*sampleShifts)[s], numantenna*numffts*sizeof(int)));
    GPUalloc += numantenna*numffts*sizeof(int);

    // Delay information vectors
    gpuErrchk(hipMalloc(&(*gpuDelays)[s], numantenna*4*sizeof(double)));
    GPUalloc += numantenna*4*sizeof(double);
  }
  
  cout << "Allocated " << GPUalloc/1e6 << " Mb on GPU" << endl;
}

void allocDataHost(uint8_t ***data, int numantenna, int subintsamples, int nbit, int nPol, bool iscomplex, int &subintbytes)
{
  subintbytes = subintsamples*nbit*nPol/8;  // Watch 31bit overflow
  cout << "Allocating " << subintbytes/1024/1024 << " MB per antenna per subint" << endl;
  cout << "           " << subintbytes * numantenna / 1024 / 1024 << " MB total" << endl;


  *data = new uint8_t*[numantenna];
  for (int a=0; a<numantenna; a++){
    gpuErrchk(hipHostMalloc(&(*data)[a], subintbytes*sizeof(uint8_t)));
  }
}

inline float carg(const hipComplex& z) {return atan2(hipCimagf(z), hipCrealf(z));} // polar angle

void saveVisibilities(const char *outfile, hipComplex *baselines, int nbaseline, int nchan, int stride, double bandwidth) {
  hipComplex **vis;
  std::ofstream fvis(outfile);

  // Copy final visibilities back to CPU
  vis = new hipComplex*[nbaseline*4];
  for (int i=0; i<nbaseline*4; i++) {
    vis[i] = new hipComplex[nchan];
    gpuErrchk(hipMemcpy(vis[i], &baselines[i*stride], nchan*sizeof(hipComplex), hipMemcpyDeviceToHost));
  }
  
  for (int c=0; c<nchan; c++) {
    fvis << std::setw(5) << c << " " << std::setw(11) << std::fixed << std::setprecision(6) << (c+0.5)/nchan*bandwidth/1e6;
    fvis  << std::setprecision(5);
    for (int i=0; i<nbaseline*4; i++) {
      fvis << " " << std::setw(11) << vis[i][c].x << " " << std::setw(11) << vis[i][c].y;
      fvis << " " << std::setw(11) << hipCabsf(vis[i][c]) << " " << std::setw(11) << carg(vis[i][c]);
    }
    fvis << std::endl;
  }
  fvis.close();
  
  for (int i=0;i<nbaseline*4;i++) {
    delete [] vis[i];
  }
  delete [] vis;
}

int main(int argc, char *argv[])
{
  // variables for the test
  char *configfile;
  int subintbytes, status, cfactor;
  int nPol;
  uint8_t ** inputdata;
  double ** delays; /**< delay polynomial for each antenna.  delay is in seconds, time is in units of FFT duration */
  double * antfileoffsets; /**< offset from each the nominal start time of the integration for each antenna data file.  
                                In units of seconds. */
  int numchannels, numantennas, nbaseline, numffts, nbit;
  double lo, bandwidth, sampletime, subinttime;
  bool iscomplex;
  vector<string> antennas, antFiles;
  vector<std::ifstream *> antStream;

  int8_t ***packedData;
  float **rotationPhaseInfo;
  float **fractionalSampleDelays;
  int **sampleShifts;
  double **gpuDelays;
  hipComplex **unpackedData, **channelisedData, **baselineData;
  hipfftHandle plan[kNumStreams];
  hipEvent_t start_exec, stop_exec;
  
  // Read in the command line arguments.
  struct arguments arguments;
  arguments.nloops = 10;
  arguments.output_binary = 0;
  arguments.configfile[0] = 0;
  argp_parse(&argp, argc, argv, 0, 0, &arguments);

  if (strlen(arguments.configfile) > 0) {
    configfile = arguments.configfile;
  }
  printf("reading configuration file %s\n", arguments.configfile);
  printf("running %d loops\n", arguments.nloops);
  printf("will output %s data\n", (arguments.output_binary == 0) ? "text" : "binary");

  hipEventCreate(&start_exec);
  hipEventCreate(&stop_exec);
  
  init_2bitLevels();

  // load up the test input data and delays from the configfile
  parseConfig(configfile, nbit, nPol, iscomplex, numchannels, numantennas, lo, bandwidth, numffts, antennas, antFiles, &delays, &antfileoffsets);
  nPol = 2;

  nbaseline = numantennas*(numantennas-1)/2;
  if (iscomplex) {
    cfactor = 1;
  } else{
    cfactor = 2; // If real data FFT size twice size of number of frequecy channels
  }

  int fftchannels = numchannels*cfactor;
  int subintsamples = numffts*fftchannels;  // Number of time samples - need to factor # channels (pols) also
  cout << "Subintsamples= " << subintsamples << endl;

  sampletime = 1.0/bandwidth;
  if (!iscomplex) sampletime /= 2.0; 
  subinttime = subintsamples*sampletime;
  cout << "Subint = " << subinttime*1000 << " msec" << endl;

  // Setup threads and blocks for the various kernels
  // Unpack
  int unpackThreads = NTHREADS;
  int unpackBlocks  = subintsamples/nPol/unpackThreads;
  if (unpackThreads*unpackBlocks*nPol!=subintsamples) {
    cerr << "Error: <<" << unpackBlocks << "," << unpackThreads << ">> inconsistent with " << subintsamples << " samples for unpack kernel" << endl;
  }

  // Fringe Rotate
  int fringeThreads, blockchan;
  if (fftchannels<=NTHREADS) {
    fringeThreads = fftchannels;
    blockchan = 1;
  } else {
    fringeThreads = NTHREADS;
    blockchan = fftchannels/NTHREADS;
    if (fftchannels%NTHREADS) {
      cerr << "Error: NTHREADS not divisible into fftchannels" << endl;
      exit(1);
    }
  }
  dim3 fringeBlocks = dim3(blockchan, numffts, numantennas);

  // Fractional Delay
  int fracDelayThreads;
  if (numchannels<=NTHREADS) {
    fracDelayThreads = numchannels;
    blockchan = 1;
  } else {
    fracDelayThreads = NTHREADS;
    blockchan = numchannels/NTHREADS;
    if (numchannels%NTHREADS) {
      cerr << "Error: NTHREADS not divisible into fftchannels" << endl;
      exit(1);
    }
  }
  dim3 fracDelayBlocks = dim3(blockchan, numffts, numantennas);

  // CrossCorr
  int targetThreads = 50e4;  // This seems a *lot*
  int corrThreads;
  if (numchannels<=512) {
    corrThreads = numchannels;
    blockchan = 1;
  } else {
    corrThreads = 512;
    blockchan = numchannels/512;
  }
  int parallelAccum = (int)ceil(targetThreads/numchannels+1); // I suspect this has failure modes
  cout << "Initial parallelAccum=" << parallelAccum << endl;
  while (parallelAccum && numffts % parallelAccum) parallelAccum--;
  if (parallelAccum==0) {
    cerr << "Error: Could not determine block size for Cross Correlation" << endl;
    exit(1);
  }
  int nchunk = numffts / parallelAccum;
  dim3 corrBlocks = dim3(blockchan, parallelAccum);
  cout << "Corr Threads:  " << corrThreads << " " << blockchan << ":" << parallelAccum << "/" << nchunk << endl;

  // Final Cross Corr accumulation
  dim3 accumBlocks = dim3(blockchan, 4, nbaseline);

  
  cout << "Allocate Memory" << endl;
  // Allocate space in the buffers for the data and the delays
  allocDataHost(&inputdata, numantennas, numchannels, numffts, nbit, nPol, iscomplex, subintbytes);

  // Allocate space on the GPU
  allocDataGPU(&packedData, &unpackedData, &channelisedData,
	             &baselineData, &rotationPhaseInfo, &fractionalSampleDelays, &sampleShifts,
               &gpuDelays, numantennas, subintsamples,
	             nbit, nPol, iscomplex, numchannels, numffts, parallelAccum, kNumStreams);

  for (int i=0; i<numantennas; i++) {
    antStream.push_back(new std::ifstream(antFiles[i].c_str(), std::ios::binary));
  }

  // Configure CUFFT
  for (int s=0; s<kNumStreams; s++) {
    if (hipfftPlan1d(&plan[s], fftchannels, HIPFFT_C2C, 2*numantennas*numffts) != HIPFFT_SUCCESS) {
      cout << "CUFFT error: Plan creation failed" << endl;
      return(0);
    }
  }
  
  cout << "Reading data" << endl;
  status = readdata(subintbytes, antStream, inputdata);
  if (status) exit(1);
  init_2bitLevels();

  // Initialise CUDA streams
  cout << "Initialising CUDA streams" << endl;
  hipStream_t streams[kNumStreams];
  for (int s=0; s<kNumStreams; s++)
    gpuErrchk(hipStreamCreate(&streams[s]));

  // Check that the number of FFTs is a valid number
  if (numffts%8)
  {
    cerr << "Error: numffts must be divisible by 8" << endl;
    exit(1);
  }
  // Set the number of blocks for fringe rotation (and fractional sample delay?)
  dim3 FringeSetblocks = dim3(8, numantennas);

  // Record the start time
  hipEventRecord(start_exec, 0);
  cout << "Entering loop" << endl;
  for (int l=0; l<arguments.nloops; l++)
  {
    int stream = l % kNumStreams;

    // Copy data to GPU
    cout << "Copy data to GPU" << endl;
    for (int i=0; i<numantennas; i++) {
      gpuErrchk(hipMemcpyAsync(packedData[stream][i], inputdata[i], subintbytes, hipMemcpyHostToDevice, streams[stream])); 
    }
    // Copy delays to GPU
    cout << "Copy delays to GPU" << endl;
    for (int i=0; i<numantennas; i++) {
      gpuErrchk(hipMemcpy(&(gpuDelays[stream][i*4]), delays[i], 3*sizeof(double), hipMemcpyHostToDevice));
      gpuErrchk(hipMemcpy(&(gpuDelays[stream][i*4+3]), &(antfileoffsets[i]), sizeof(double), hipMemcpyHostToDevice));
    }

    // Use the delays to calculate fringe rotation phases and fractional sample delays for each FFT //
    calculateDelaysAndPhases<<<FringeSetblocks, numffts/8>>>(gpuDelays[stream], lo, sampletime, fftchannels, numchannels, rotationPhaseInfo[stream], 
                                                             sampleShifts[stream], fractionalSampleDelays[stream]);
    CudaCheckError();

    // Unpack the data
    //cout << "Unpack data" << endl;
    for (int i=0; i<numantennas; i++) {
      unpack2bit_2chan_fast<<<unpackBlocks,unpackThreads,0,streams[stream]>>>(&unpackedData[stream][2*i*subintsamples], packedData[stream][i], &(sampleShifts[stream][numffts*i]));
      CudaCheckError();
    }

    // Fringe Rotate //
    cout << "Fringe Rotate" << endl;
    setFringeRotation<<<FringeSetblocks, numffts/8,0,streams[stream]>>>(rotationPhaseInfo[stream]);
    CudaCheckError();

    FringeRotate<<<fringeBlocks,fringeThreads,0,streams[stream]>>>(unpackedData[stream], rotationPhaseInfo[stream]);
    CudaCheckError();
  
    // FFT
    cout << "FFT data" << endl;
    hipfftSetStream(plan[stream], streams[stream]);
    if (hipfftExecC2C(plan[stream], unpackedData[stream], channelisedData[stream], HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
      cout << "CUFFT error: ExecC2C Forward failed" << endl;
      return(0);
    }

    // Fractional Delay Correction
    //FracSampleCorrection<<<fracDelayBlocks,fracDelayThreads>>>(channelisedData, fractionalDelayValues, numchannels, fftchannels, numffts, subintsamples);
    //CudaCheckError();
    
    // Cross correlate
    cout << "Cross correlate" << endl;
    gpuErrchk(hipMemsetAsync(baselineData[stream], 0, nbaseline*4*numchannels*parallelAccum*sizeof(hipComplex), streams[stream]));

#if 0
    CrossCorr<<<corrBlocks,corrThreads,0,streams[stream]>>>(channelisedData[stream], baselineData[stream], numantennas, nchunk);
    CudaCheckError();
    // cout << "Finalise" << endl;
    finaliseAccum<<<accumBlocks,corrThreads,0,streams[stream]>>>(baselineData[stream], parallelAccum, nchunk);
    CudaCheckError();
#else
    int ccblock_width = 128;
    dim3 ccblock(1+(numchannels-1)/ccblock_width, numantennas-1, numantennas-1);
    CrossCorrAccumHoriz<2><<<ccblock, ccblock_width, 0, streams[stream]>>>(baselineData[stream], channelisedData[stream], numantennas, numffts, numchannels, fftchannels);
#endif
  }
  
  float dtime;
  hipEventRecord(stop_exec, 0);
  hipEventSynchronize(stop_exec);
  hipEventElapsedTime(&dtime, start_exec, stop_exec);

  cout << "Total execution time for " << arguments.nloops << " loops =  " <<  dtime << " ms" << endl;

// I have chosen to write out the first streams data, this will need to be thought about more carefully
#if 0
  saveVisibilities("vis.out", baselineData[0], nbaseline, numchannels, parallelAccum*numchannels, bandwidth);
#else
  saveVisibilities("vis.out", baselineData[0], nbaseline, numchannels, numchannels, bandwidth);
#endif

  hipDeviceSynchronize();
  hipDeviceReset();

  // Calculate the elapsed time

  // Free memory
  //  for (i=0; i<numantennas; i++)
  //{
  //  delete(inputdata[i]);
  //  delete(delays[i]);
  //}
  //delete(inputdata);
  //delete(delays);
}
